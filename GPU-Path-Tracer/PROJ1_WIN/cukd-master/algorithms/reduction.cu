#include "hip/hip_runtime.h"
// Copyright (c) 2012, Thomas Schutzmeier
// FreeBSD License
//
// This software contains source code (or derivatives) provided by
// NVIDIA Corporation. See LICENSE for details.
//
// See https://github.com/unvirtual/cukd/blob/master/LICENSE

#include <iostream>
#include "algorithms/shared_mem.h"
#include "algorithms/reduction.h"
 
/**********************************************************************************
 *
 * Kernels
 *
 **********************************************************************************/

template <unsigned int chunk_size, typename T, class Method>
__global__
void chunk_reduction_kernel(T* element_list,
                            T* output,
                            int* first_index_list,
                            int* chunk_lengths) {

    T* sarr = SharedMemory<T>();
    unsigned int chunk_index = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int first_index, chunk_length;

    first_index = first_index_list[chunk_index];
    chunk_length = chunk_lengths[chunk_index];
    __syncthreads();
    if (tid < chunk_length) {
        sarr[tid] = element_list[tid + first_index];
    } else {
        sarr[tid] = Method::neutral_element();
    }
    __syncthreads();

    T result = reduction_device<chunk_size, T, Method>(sarr);

    if(tid == 0) {
        output[chunk_index] = result;
    }
}

template<typename T, class Method>
__global__
void segmented_reduction_kernel(T* values, int n_values, int* keys, int n_keys,
                                int* keyranges, T* result) {
    __shared__ int skeys[256];
    __shared__ int svalues[256];
    T* sres = SharedMemory<T>();
    int minkey = keyranges[2*blockIdx.x];
    int keydiff = keyranges[2*blockIdx.x + 1] - minkey;

    int thread = threadIdx.x;
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    // load keys and values
    svalues[thread] = values[index];
    skeys[thread] = keys[index];

    // result with the proper length
    if(thread <= keydiff)
        sres[thread] = 0;

    __syncthreads();
    for(int i = 1; i < blockDim.x; i *= 2) {
        if(thread % (2*i) == 0) {
            int w0 = skeys[thread];
            int w1 = skeys[thread + i];
            if(w0 != w1) {
                sres[w1 - minkey] += svalues[thread + i];
            }
            else {
                svalues[thread] += svalues[thread + i];
            }
        }
        __syncthreads();
    }
    // atomicAdd is fine here, as there are only few of those ops per
    // thread
    if(thread <= keydiff)
        atomicAdd(&result[minkey+thread], sres[thread]);
    __syncthreads();
    if(thread == 0)
        atomicAdd(&result[skeys[0]],svalues[0]);
}

// Prepare keys for segmented reduction
void __global__ seg_reduce_keyrange_per_block(int* keys, int* ranges,
                                              int n, int blockdim) {
    int i = threadIdx.x;
    if(i < blockdim) {
        ranges[2*i] = keys[i*blockdim];
        ranges[2*i+1] = keys[(i+1)*blockdim - 1];
    }
}

/**********************************************************************************
 *
 * Kernel Wrappers
 *
 **********************************************************************************/

template<unsigned int chunk_size, typename T, class Method>
void
chunk_reduce(T* element_list, T* output, int n_chunks,
             int* first_index_list, int* chunk_lengths) {
    dim3 blocks = dim3(chunk_size, 1, 1);
    dim3 grid = dim3(n_chunks, 1, 1);
    int shared_size = chunk_size*sizeof(T);

    chunk_reduction_kernel<chunk_size, T, Method> <<<grid,blocks,shared_size>>>
        (element_list, output, first_index_list, chunk_lengths);
}

template<typename T, class Method>
void
segmented_reduce(T* values, int n_values, int* keys, int n_keys, T* result) {
    int n_threads = 256;
    int n_blocks = n_values/n_threads;
    int* dev_keyranges;
    cutilSafeCall(hipMalloc((void**) &dev_keyranges, 2*n_blocks*sizeof(int)));

    dim3 dim_block_ranges(n_blocks,1,1);
    dim3 dim_grid_ranges(1,1,1);
    seg_reduce_keyrange_per_block<<<dim_grid_ranges, dim_block_ranges>>>(
            keys, dev_keyranges, n_values, n_threads);

    dim3 dim_block(n_threads, 1, 1);
    dim3 dim_grid(n_blocks, 1, 1);
    int smesize = n_threads*3*sizeof(int);
    segmented_reduction_kernel<T,Method><<<dim_grid, dim_block,smesize>>> (
            values, n_values, keys, n_keys, dev_keyranges, result);
    CUT_CHECK_ERROR("Segmented Reduction kernel call failed");
    hipFree(dev_keyranges);
}

/**********************************************************************************
 *
 * Template specializations
 *
 **********************************************************************************/

template void
chunk_reduce<256, int, SumReductionMethod<int> >(
        int* element_list, int* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, float, SumReductionMethod<float> >(
        float* element_list, float* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, int, MinReductionMethod<int> >(
        int* element_list, int* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, int, MaxReductionMethod<int> >(
        int* element_list, int* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, float, MinReductionMethod<float> >(
        float* element_list, float* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, float, MaxReductionMethod<float> >(
        float* element_list, float* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, float4, MinReductionMethod<float4> >(
        float4* element_list, float4* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, float4, MaxReductionMethod<float4> >(
        float4* element_list, float4* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, UFloat4, MinReductionMethod<UFloat4> >(
        UFloat4* element_list, UFloat4* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
chunk_reduce<256, UFloat4, MaxReductionMethod<UFloat4> >(
        UFloat4* element_list, UFloat4* output, int n_chunks,
        int* first_index_list, int* chunk_lengths);

template void
segmented_reduce<int, SumReductionMethod<int> >(int* values, int n_values, int* keys,
                                                int n_keys, int* result);
