#include "hip/hip_runtime.h"
// Copyright (c) 2012, Thomas Schutzmeier
// FreeBSD License
// See https://github.com/unvirtual/cukd/blob/master/LICENSE

#include "small_node_array.h"

namespace cukd {
namespace device {
__global__
void
compute_SAH_kernel(device::SmallNodeArray active, device::SplitCandidateArray sca,
                   int* min_sah_split_idx, float* min_sah_cost);

} // namespace device


void
SmallNodeArray::best_split_SAH(SplitCandidateArray & sca,
                               DevVector<int> & min_sah_split_pos,
                               DevVector<float> & min_sah_cost) {
    dim3 grid(n_nodes(), 1,1);
    dim3 blocks(6*64,1,1);
    device::compute_SAH_kernel<<<grid, blocks>>>(dev_array(), sca.dev_array(),
                                                 min_sah_split_pos.pointer(),
                                                 min_sah_cost.pointer());
    CUT_CHECK_ERROR("compute_SAH_kernel failed");
}

} // cukd
